#include "hip/hip_runtime.h"
//============================================================================
// Name        : interface_for_simple_data_structures.cpp
// Author      : Christos Tsotskas
// Version     :
// Copyright   : 
// Description : Hello World in C++, Ansi-style
//============================================================================

#include <iostream>
#include <map>
#include <vector>
#include <cmath>



#include "test.cuh"


#ifdef GPU_evaluation


void CUDA_evaluation::strip_to_1D(const Container2 &in_map, double* out_decision_variables, double* out_objectives){
#define var_index_map(entry_index,var_index) (entry_index*nVar+var_index)
#define obj_index_map(entry_index,obj_index) (entry_index*nObj+obj_index)

	const unsigned int nEntries=in_map.size() ;
	const unsigned int nVar=(in_map.begin())->first.size() ;
	const unsigned int nObj=(in_map.begin())->second.size() ;

	unsigned int entry_index=0;
	for(std::map< std::vector<double>, std::vector<double> >::const_iterator  it=in_map.begin(); it!=in_map.end(); ++it){
		for(unsigned int var_index=0 ; var_index<nVar; ++var_index)
			//			out_decision_variables[entry_index*nVar+var_index]=it->first[var_index];
			out_decision_variables[var_index_map(entry_index,var_index)]=it->first[var_index];

		for(unsigned int obj_index=0 ; obj_index<nObj; ++obj_index)
			//			out_objectives[entry_index*nObj+obj_index]=it->second[obj_index];
			out_objectives[obj_index_map(entry_index,obj_index)]=it->second[obj_index];

		++entry_index;
	}

	//	int a;
	//	std::cout << "type int" << std::endl;
	//	std:: cin >> a;
#undef var_index_map
#undef obj_index_map
}

void CUDA_evaluation::display_data(const double* dataset,const int entries, const int col){
	for(int i=0; i<entries; ++i){
		std::cout << i  << ": ";
		for(int j=0; j<col; ++j){
			std::cout << dataset[i*col+j] << " ";
		}
		std::cout  << std::endl;
	}
}


void CUDA_evaluation::display_data(const std::map< std::vector<double>, std::vector<double> > &in_map){
	int entry=0;
	for(std::map< std::vector<double>, std::vector<double> >::const_iterator  it=in_map.begin(); it!=in_map.end(); ++it){
		std::cout << entry  << ": ";

		for(unsigned int var_index=0 ; var_index<it->first.size(); ++var_index)
			std::cout << it->first[var_index] << " ";

		std::cout << "===>";
		for(unsigned int obj_index=0 ; obj_index<it->second.size(); ++obj_index)
			std::cout << it->second[obj_index] << " ";


		std::cout  << std::endl;
		++entry;
	}
}

double CUDA_evaluation::calc_obj1(const double &a, const double &b){
	return a+b;
}

double CUDA_evaluation::calc_obj2(const double& a, const double& b){
	return a+b;
}
//The design of the optimiser was based on a different basis, compared to the traditional approach.
//More specifically, there are two key differences. First, the algorithm was designed to match the capabilities of the hardware
// and it can scale by adding a more powerful in terms of specifications hardware. In turn, the main principle is to fully utilise
// the available computational resources as much as possible.
//Second, it aims to tackle multi-objective
//problems and with some trivial tweaking single-objective problems can be dealt, too. Consequently, the internal data structures
//and algorithms were selected to implement the Pareto-optimality satisfactorily.

// Since the hardware is not mature enough, it is expected not to be immediately applicable. However, as the technology becomes
//more matures, this approach will tend to be more realistic.


//extra, following the locality of data, there could be re-usability of some components, if the blocks seems similar

//as an extension, the code, could separate the list of samples on blocks, based on similarity patterns (and would also inform the GPU about the similarity).
//This is done to avoid any unexpected searchs and branches at the execution.

//as an extension to the structure of the optimiser, the pattern move should be altered to be more collective, not sequential
//as it is currently.

//as an extentsion, currently all the objectives of a single design are evaluated by a single thread. It worths to investigate how the code performs when each objective is evaluated on a separate thread (which could simplify/alleviate the computational requirements)

//as an extension, the time required to send and receive data from GPU can be improved by using the stream feature from CUDA. Although this was not used here, it is expected to shortent the exchange time when the number of variables grows significantly.

//at conclusions, the presented approch for GPU implementation was selected so as to allow for future implementations to use GPUs' spatial locality more efficiently.

//idea: map an entry to a thread, the total number of threads launched equals the sampling. however, it is not guaranteed that
//this number will always be the same. The GPU can be used in many different ways. Here, the selected approach is to do every evaluation
//on each thread, or simply it serves as an evaluation-factory. Each evaluation is independent from the other, however they are evaluated
// in batches.
//(so it can just scale up as the more resources are added, one requirement is that the problem should ask/produce so many points to be evaluated.).
//The CPU part will do the high level and complex part of the code, the evaluations
void CUDA_evaluation::run_thread(unsigned int thread,
		unsigned int block,
		const unsigned int nEntries,
		const double* out_decision_variables,
		const unsigned int nVar,
		double* out_objectives,
		const unsigned int nObj){

#define var_index_map(entry_index,var_index) (entry_index*nVar+var_index)
#define obj_index_map(entry_index,obj_index) (entry_index*nObj+obj_index)

	const unsigned int Nthreads=2;
	const unsigned int gid=block* Nthreads+thread;

	if(gid<nEntries){

		out_objectives[obj_index_map(gid,0)]=calc_obj1(out_decision_variables[var_index_map(gid,0)], out_decision_variables[var_index_map(gid,1)]);

		out_objectives[obj_index_map(gid,1)]=calc_obj2(out_decision_variables[var_index_map(gid,1)], out_decision_variables[var_index_map(gid,2)]);

	}
	std::cout<<"gid:" << gid << " calculated:" << out_objectives[obj_index_map(gid,0)] << " and " << out_objectives[obj_index_map(gid,1)] << std::endl;

#undef var_index_map
#undef obj_index_map
}

void CUDA_evaluation::run() {

#ifdef DEBUG
	std::cout << "testing_kernel" << std::endl;
#endif //DEBUG	
	calc_objectives<<<number_of_blocks, number_of_threads>>>(entries, dev_variables, dev_objective, nVar, nObj);
	hipDeviceSynchronize();
	receive_from_GPU();
//	link_objectives_to_map();

}

void CUDA_evaluation::allocate_memory(){
	hipMalloc( (void**) &dev_variables ,  nVar_total_size);
	hipMalloc( (void**) &dev_objective ,  nObj_total_size);
}

void CUDA_evaluation::send_to_device(){
	hipMemcpy( dev_variables, decision_variables_h, nVar_total_size , hipMemcpyHostToDevice );
	//hipMemcpy( dev_objective, objectives_h, nObj_total_size , hipMemcpyHostToDevice );
}




CUDA_evaluation::CUDA_evaluation(int selected_threads, const Container2& temp_list)
				{

	entries= temp_list.size();
	nVar=(temp_list.begin())->first.size();
	nObj=(temp_list.begin())->second.size();

	decision_variables_h= new double[ entries * nVar ];
	objectives_h= new double[ entries * nObj ];

	number_of_threads=selected_threads;
	number_of_blocks= (int) std::ceil (1.0*entries/number_of_threads*1.0);

	std::cout << "received:" << entries << " entries" << std::endl;
	std::cout << "number_of_threads:" << number_of_threads << std::endl;
	std::cout << "number_of_blocks:" << number_of_blocks << std::endl;
#ifdef DEBUG
	temp_list.displayContainer();
#endif //DEBUG
	strip_to_1D(temp_list, decision_variables_h, objectives_h);

	nVar_total_size=entries *nVar*sizeof(double);
	nObj_total_size=entries *nObj*sizeof(double);

#ifdef DEBUG
	std::cout << "variables' size "<< entries << "x" << nVar << std::endl;
	std::cout << "objectives' size "<< entries << "x" << nObj << std::endl;

	std::cout << "received variables" << std::endl;
	display_data(decision_variables_h, entries, nVar);

	std::cout << "received objectives" << std::endl;
	display_data(objectives_h, entries, nObj);
#endif //DEBUG

	allocate_memory();
	send_to_device();
#ifdef DEBUG
	display_data(temp_list);
#endif //DEBUG
}

__global__ void internal_print_variables(const int entries, const double* variables, const double* objectives, const int nVar, const int nObj){

#define var_index_map(entry_index,var_index) (entry_index*nVar+var_index)
#define obj_index_map(entry_index,obj_index) (entry_index*nObj+obj_index)

	int gid=threadIdx.x + blockIdx.x*blockDim.x;


	if(gid<entries){
		printf("cuda_gid %d's variables: %f %f %f\n", gid, variables[var_index_map(gid,0)], variables[var_index_map(gid,1)], variables[var_index_map(gid,2)]);
	}
	__syncthreads();
#undef var_index_map
#undef obj_index_map
}


__global__ void internal_print_objectives(const int entries, const double* variables, const double* objectives, const int nVar, const int nObj){

#define var_index_map(entry_index,var_index) (entry_index*nVar+var_index)
#define obj_index_map(entry_index,obj_index) (entry_index*nObj+obj_index)

	int gid=threadIdx.x + blockIdx.x*blockDim.x;
	if(gid<entries){
		printf("cuda_gid %d's objectives: %f %f\n", gid, objectives[obj_index_map(gid,0)], objectives[obj_index_map(gid,1)] );
	}
	__syncthreads();
#undef var_index_map
#undef obj_index_map
}

__device__ double objective2(const double* variables, const int nVar, const double f1, const int entry) {
#define var_index_map(entry_index,var_index) (entry_index*nVar+var_index)
	double g = 0.0;
	double h = 0.0;

	for (int i = 1; i < nVar; i++){
		g += variables[var_index_map(entry,i)];
	}
	g = 1.0 + 9.0 * g / (nVar-1);
	h = 1.0 - std::pow(f1 / g, 2);

#undef var_index_map
	return g * h;
}

__global__ void calc_objectives(const int entries, const double* variables, double* objectives, const int nVar, const int nObj){

#define var_index_map(entry_index,var_index) (entry_index*nVar+var_index)
#define obj_index_map(entry_index,obj_index) (entry_index*nObj+obj_index)

	int gid=threadIdx.x + blockIdx.x*blockDim.x;
	if(gid<entries){

		const double f1 = variables[var_index_map(gid,0)];

		objectives[obj_index_map(gid,0)]=f1;
		objectives[obj_index_map(gid,1)]= objective2(variables, nVar, f1, gid);
#ifdef DEBUG
		printf("gid:%d, calculated obj1: %f and obj2:%f \n",gid, objectives[obj_index_map(gid,0)], objectives[obj_index_map(gid,1)] );
#endif //DEBUG
	}

	__syncthreads();
#undef var_index_map
#undef obj_index_map	
}


void CUDA_evaluation::print_CUDA(){




	internal_print_variables<<<number_of_blocks, number_of_threads>>>(entries, dev_variables, dev_objective, nVar, nObj);
	internal_print_objectives<<<number_of_blocks, number_of_threads>>>(entries, dev_variables, dev_objective, nVar, nObj);	

}
void CUDA_evaluation::receive_from_GPU(){
	hipMemcpy( objectives_h, dev_objective, nObj_total_size , hipMemcpyDeviceToHost );
}


void CUDA_evaluation::check_results(){

	std::vector<double> temp_check_objective(nObj,0.0 );
	/*for(std::map< std::vector<double>, std::vector<double> >::iterator it=temp_list.begin(); it!=temp_list.end(); ++it ){

		for(int obj=0; obj<nObj; ++obj)
			for(int var=0; var<nVar; ++var)
				temp_check_objective[obj]=2;
	}*/
}

void CUDA_evaluation::update_container_objectives(Container2& __temp_list){

	int entry=0;
	ObjFunction2 temp_buffer(nObj,5555.0);
	for(Container2::iterator  it=__temp_list.begin(); it!=__temp_list.end(); ++it){
		for(unsigned int obj_index=0 ; obj_index<nObj; ++obj_index){
			temp_buffer[obj_index]=objectives_h[entry];
			++entry;
		}
		it->second=temp_buffer;
	}



}


//void CUDA_evaluation::link_objectives_to_map(){
//
//	hipDeviceSynchronize();
//	int entry=0;
//	ObjFunction2 temp_buffer(nObj,5555.0);
//	for(Container2::iterator  it=temp_list.begin(); it!=temp_list.end(); ++it){
//		for(unsigned int obj_index=0 ; obj_index<nObj; ++obj_index){
//			temp_buffer[obj_index]=objectives_h[entry];
//			++entry;
//		}
//		it->second=temp_buffer;
//	}
//	hipDeviceSynchronize();
//	std::cout << "within the mapping" << std::endl;
//	temp_list.displayContainer();
//}


CUDA_evaluation::~CUDA_evaluation(){
#ifdef DEBUG
	print_CUDA();
#endif //DEBUG

#ifdef DEBUG
	std::cout << "finally, variables" << std::endl;
	display_data(decision_variables_h, entries, nVar);

	std::cout << "finally, objectives" << std::endl;
	display_data(objectives_h, entries, nObj);
#endif //DEBUG
	cutilSafeCall(hipFree(dev_variables));
	cutilSafeCall(hipFree(dev_objective));



	free(decision_variables_h);
	free(objectives_h);

	std::cout  << "CUDA evaluations has just finished and evaluated " << entries<<" solutions. Memories were freed." << std::endl;
}



#endif //GPU_evaluation


